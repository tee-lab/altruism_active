#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

 /* This example demonstrates how to use the Cuda OpenGL bindings with the
  * runtime API.
  * Device code.
  */

#ifndef _KERNELS_CU_
#define _KERNELS_CU_

#include <hiprand/hiprand_kernel.h>
#include "params.h"
#include "globals.h"
#include "utils/cuda_vector_math.cuh"
#include "utils/cuda_device.h"
#include <thrust/scan.h>

#include "utils/simple_io.h"

// simulation parameters in constant memory
__constant__ SimParams params;

hipError_t __host__ copyParams(SimParams *s){
	return hipMemcpyToSymbol(HIP_SYMBOL(params), s, sizeof(SimParams));
}


// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// KERNEL to set up RANDOM GENERATOR STATES
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
__global__ void rngStateSetup_kernel(int * rng_Seeds, hiprandState * rngStates){
	//int tid = threadIdx.x;							// each block produces exactly the same random numbers
	int tid_u = threadIdx.x + blockIdx.x*blockDim.x;	// each block produces different random numbers
	
	hiprand_init (rng_Seeds[tid_u], 0, 0, &rngStates[tid_u]);
}


// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// KERNELs to find NEAREST NEIGHBOURS using GRID
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

void exclusive_scan(int * in, int * out, int n){
	out[0]=0;
	for (int i=1; i<n; ++i){
		out[i] = out[i-1]+in[i-1];
	}
}

// given the position, get the cell ID on a square grid of dimensions nxGrid x nxGrid,
// with each cell of size cellSize
// this function returns cellId considering 0 for 1st grid cell. With multiple blocks, user must add appropriate offset
//		|---|---|---|---|---|
//		|   |   |   |   |   |
//		|---|---|---|---|---|
//		|   |   | x |   |   |	<-- x = (pos.x, pos.y)
//		|---|---|---|---|---|
//		|   |   |   |   |   |
//		|---|---|---|---|---|
//      ^ 0 = (xmin, ymin)	^ nx = xmin + nx*cellWidth

inline __device__ int getCellId(float2 pos, int nxGrid, int cellwidth){//, SimParams *s){
	int ix = (pos.x-params.xmin)/(cellwidth+1e-6);	// add 1e-6 to make sure that particles on edge of last cell are included in that cell
	int iy = (pos.y-params.ymin)/(cellwidth+1e-6);
	return iy*nxGrid + ix;
}

// calculate cellID for each particle and count # particles / cell
// while storing, full cellID is stored 
// This kernel MUST BE launched with <<< nBlocks x nFish >>> config.
//  ^ this constraint is kept for intuitive reasons. To remove it, use pid/np in place of blockIdx.x
__global__ void gridCount_kernel(float2 * pos_array, int * cellId_array, int * gridCount_array, int nxGrid, int _cellSize, /*SimParams *s,*/ const int np, const int nb){
	unsigned int pid = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (pid < np*nb){
		int cellId_p = getCellId(pos_array[pid], nxGrid, _cellSize);//, s);	// get principle cell Id of particle pid
		int cellId = ix2(cellId_p, blockIdx.x, nxGrid*nxGrid);			// grid dimension is nCells x nBlocks
		atomicAdd(&gridCount_array[cellId],1);					// gridCount must be addressed using full cellId
	
		//++gridCount_array[pid];
		cellId_array[pid] = cellId;		// cellIds array stores principle cellId
	}
}

// rewrite particles in blockwise sorted order using results of scan
// full particle IDs are written in sorted order
// This kernel MUST BE launched with <<< nBlocks x nFish >>> config
__global__ void sortParticles_kernel(int* cummCount_array, int *filledCount_array, int * cellIds_array, int *sortedIds_array, int nxGrid, const int np, const int nb){

	unsigned int pid = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (pid < np*nb){
		int cell = cellIds_array[pid];
		//int cell = ix2(cell_p, blockIdx.x, nxGrid*nxGrid);
		int sortedAddress_p = cummCount_array[cell] + atomicAdd(&filledCount_array[cell],1); // atomicAdd returns old value
		int sortedAddress = ix2(sortedAddress_p, blockIdx.x, np);	// use of blockIdx.x here necessitates launching with  <<< nBlocks x nFish >>> config
		sortedIds_array[sortedAddress] = pid;
	}
}

// this kernel gets the start and end ids of particles in each grid cell. 
// Hence it must be run using 1 thread for each grid cell = nBlocks*nCells threads
// startId and endId arrays store full particle IDs. 
// This kernel MUST BE launched with <<< nBlocks x nFish >>> config
__global__ void getParticleIds_kernel(int * pStartIds_array, int* pEndIds_array, int * cellIds_array, int* cummCount_array, int* gridCount_array, const int np, const int nb){
	unsigned int pid = blockIdx.x*blockDim.x + threadIdx.x;

	if (pid < np*nb){
		int gid = cellIds_array[pid];		// full pid automatically gives full gid
		int count = gridCount_array[gid];	// number of particles in cell gid
		int scan = cummCount_array[gid];	// cummulative number of particles till cell gid in block blockIdx.x
		
		int startId = scan*float(count!=0) + -1*float(count==0);	// if count is zero, set startId to -1
		int endId = startId + count-1;	// if count is zero, this will become -2. This is FINE.

		pStartIds_array[gid] = ix2(startId, blockIdx.x, np);
		pEndIds_array[gid]   = ix2(  endId, blockIdx.x, np);
	}
}


// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// KERNEL TO EXECUTE MOVEMENT OF A SINGLE FISH
// This is a CRACKING KERNEL because ALL conditionals 
// are replaced with indicator variables !!! :) :)
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
__global__ void movement_kernel(float2* pos_array, float2* vel_array, float* Rs_array, 
								int* cellIds_array, int* sortedIds_array, 
								int* pStartIds_array, int* pEndIds_array, int nxGrid, int nGrid,
								unsigned int nfish, //SimParams * dev_params,
								hiprandState * RNGstates, float* kA_array, float* kO_array){
	
	unsigned int myID = blockIdx.x*blockDim.x + threadIdx.x;	// full particle ID. Principle particle ID is used only for accessing shared memory

	extern __shared__ float sharedMem[];
	float2 * pos_all = (float2*) sharedMem;
	float2 * vel_all = (float2*) &pos_all[nfish];
	float  * Rs_all  = (float*)  &vel_all[nfish];
	
	// copy self data from global memory to shared memory and wait for all threads to finish
	pos_all[threadIdx.x] = pos_array[myID];	// pos array has positions at alternate locations
	vel_all[threadIdx.x] = vel_array[myID];
	 Rs_all[threadIdx.x] =  Rs_array[myID]; 
	
	// init directions of attraction and repulsion. Only one of these will be considered
	float2 dirR = make_float2(0,0);	// initialize direction of repulsion with 0 norm
	float2 dirA = make_float2(0,0);	// initialize direction of attraction with 0 norm
	float2 dirO = make_float2(0,0);	// initialize direction of orientation with 0 norm

	__syncthreads();


	// local copies of my pos and vel. These will be modified.
//	float2 myPos = pos_all[threadIdx.x];
//	float2 myVel = vel_all[threadIdx.x];
	#define myPos (pos_all[threadIdx.x])
	#define myVel (vel_all[threadIdx.x])


	int myCell = cellIds_array[myID] % nGrid;	// get priciple cellId of focal particle. Note: myID is full particle address
	int myCellx = myCell % nxGrid;		// convert grid cell to x and y indices
	int myCelly = myCell / nxGrid;

	// loop over NINE neighbouring cells (including own cell) to find particles
	for (int innx=-1; innx<2; ++innx){			//  offsets to add in x and y indices to get neighbour cells
		for (int inny=-1; inny<2; ++inny){	

			// periodify the neighboring cell and get its address. The following code is equivalent to
			// 	 otherCellx = myCellx + innx; if (otherCellx < 0) otherCellx += nxGrid; if (otherCellx >= nxGrid) otherCellx -= nxGrid;
			int otherCellx = myCellx + innx;
			otherCellx = otherCellx + int(otherCellx < 0)*nxGrid - int(otherCellx >= nxGrid)*nxGrid;
			int otherCelly = myCelly + inny;
			otherCelly = otherCelly + int(otherCelly < 0)*nxGrid - int(otherCelly >= nxGrid)*nxGrid;

			int otherCell = ix2(otherCellx, otherCelly, nxGrid); 		// calculate principle cellId from (x,y) 
			int otherCellFull = ix2(otherCell, blockIdx.x, nGrid);		// get full cellId 

			// loop across particles found in the cell
			int start = pStartIds_array[otherCellFull];
			int end   = pEndIds_array[otherCellFull];
			for (int si=start; si <=end; ++si){ // si is particle index in sorted array
				int i = sortedIds_array[si] % nfish;	// get principle particle index from sorted particles array - Note that sorted array gives full particle ID
				if (i == threadIdx.x) continue;			// Exclude self
			
				// get direction and distance to other 
				float2 v2other = periodicDisplacement(	myPos, pos_all[i], 
														params.xmax-params.xmin, 
														params.ymax-params.ymin  );
				float d2other = length(v2other);
		
				// indicator variables 
				float Irr = float(d2other < params.Rr); //? 1:0;
				float Ira = float(d2other < Rs_all[threadIdx.x]); //? 1:0;
		
				// keep adding to dirR and dirA so that average direction or R/A will be taken
				v2other = normalize(v2other); // normalise to consider direction only

				dirR = dirR - v2other*Irr;				// add repulsion only if other fish lies in inside Rr
				dirA = dirA + v2other*Ira*(1-Irr); 		// add attraction only if other fish lies in (Rr < r < Ra)
				dirO = dirO + vel_all[i]*Ira*(1-Irr);	// add alignment only if other fish lies in (Rr < r < Ra)
			}
		}
	}

	// calculate direction of orientation (either unit or zero)
	dirO = normalizeSafeZero(dirO);
	dirA = normalizeSafeZero(dirA);

	
	// calculate direction of social interaction
	float Ir = float(length(dirR) > 1e-6);	// fish in Rr
	float Ia = float(length(dirA) > 1e-6);	// fish in Ra and hence also in Rr

	float2 dirS = myVel*(1-params.kA-params.kO) + (dirA*params.kA + dirO*params.kO);	
	//dirS = normalize(dirS);


	// final direction is either of the three terms below:
	// 1 - previous direction if no one is in either Ra or Rr (Ia = Ir = 0)
	// 2 - only repulsion direction if someone is in Rr (Ir = 1, Ia = 1)
	// 3 - social direction if someone is in Ra but no one in Rr (Ir = 0, Ia = 1)
	float2 finalDir = myVel*(1-Ir)*(1-Ia) + dirR*Ir + dirS*Ia*(1-Ir);	// dir is guaranteed to be non-zero
	finalDir = normalize(finalDir);
	
	// introduce error in following direction
	finalDir += hiprand_normal2(&RNGstates[myID])*params.errSd; 
	finalDir = normalize(finalDir);


	// impose a turning rate constraint
	float sinT = myVel.x*finalDir.y - myVel.y*finalDir.x;		// sinT = myVel x finalDir
	float cosT = dot(finalDir, myVel);	// Desired turning angle. Both vectors are unit so dot product is cos(theta) 
	float cosL = clamp( max(cosT, params.cosphi), -1.f, 1.f);
	float sinL = sqrtf(1-cosL*cosL);
	sinL = sinL - 2*sinL*float(sinT < 0);	// equivalent to: if (sinT < 0) sinL = -sinL;
	float2 a = make_float2(myVel.x*cosL - myVel.y*sinL, myVel.x*sinL + myVel.y*cosL);

	// no writing to shared memory has happened till here.
	// wait for all threads to finish their work in (reading from) shared memory. Then update shared memory
	__syncthreads();

	myVel = normalize(a);		// final velocity 
	myPos = myPos + myVel * (params.speed * params.dt);	
	makePeriodic(myPos.x, params.xmin, params.xmax);
	makePeriodic(myPos.y, params.ymin, params.ymax);
	
	// wait for all threads to update shared memory, then update global memory
	__syncthreads();


	// update pos and vel in global memory
	pos_array[myID] = myPos; 
	vel_array[myID] = myVel;
//	RNGstates[myID] = localState;
	
}

void print_devArray(int * vdev, int n){
	int * v = new int[n];
	hipMemcpy(v, vdev, n*sizeof(int), hipMemcpyDeviceToHost);
	printArray(v,n);
	delete [] v;
}

// Launcher for movement kernel - Uses many global variables
void launch_movement_kernel(){


	// reset counting arrays to Zero
	thrust::fill( (thrust::device_ptr <int>)gridCount_dev,   (thrust::device_ptr <int>)gridCount_dev   + nCells*nBlocks, (int)0);
	thrust::fill( (thrust::device_ptr <int>)filledCount_dev, (thrust::device_ptr <int>)filledCount_dev + nCells*nBlocks, (int)0);
	
//	cout << "GRIDCOUNT\n";
//	print_devArray(gridCount_dev, nCells*nBlocks);

	// count particles / grid cell
	gridCount_kernel <<<nBlocks, nFish >>> (pos_dev, cellIds_dev, gridCount_dev, nCellsX, cellSize, /*dev_params,*/ nFish, nBlocks);
	getLastCudaError("Grid Count");

//	cout << "GRIDCOUNT\n";
//	print_devArray(gridCount_dev, nCells);

	// scan - calc cummulative particles/cell
	hipMemcpy(gridCount, gridCount_dev, nBlocks*nCells*sizeof(int), hipMemcpyDeviceToHost);
	for (int iblock=0; iblock<nBlocks; ++iblock){
//		thrust::device_ptr <int> v_in  = (thrust::device_ptr <int>) &gridCount_dev[ix2(0,iblock,nCells)];
//		thrust::device_ptr <int> v_out = (thrust::device_ptr <int>) &cummCount_dev[ix2(0,iblock,nCells)];
//		thrust::exclusive_scan(v_in, v_in+nCells, v_out);
		exclusive_scan(&gridCount[ix2(0,iblock,nCells)], &cummCount[ix2(0,iblock,nCells)], nCells);
	}
	hipMemcpy(cummCount_dev, cummCount, nBlocks*nCells*sizeof(int), hipMemcpyHostToDevice);
	

//	cout << "CUMMCOUNT\n";
//	print_devArray(cummCount_dev, nCells*nBlocks);

	// sort particles by grid ID - using scan results
	sortParticles_kernel <<<nBlocks, nFish >>> (cummCount_dev, filledCount_dev, cellIds_dev, sortedIds_dev, nCellsX, nFish, nBlocks);

//	cout << "SORTEDIDS\n";
//	print_devArray(sortedIds_dev, nFish*nBlocks);

	// set start and end Ids of grids with no particles to -1 and -2 respectively
	thrust::fill((thrust::device_ptr <int>)pStartIds_dev, (thrust::device_ptr <int>)pStartIds_dev+nCells*nBlocks, (int)-1);
	thrust::fill((thrust::device_ptr <int>)pEndIds_dev,   (thrust::device_ptr <int>)pEndIds_dev  +nCells*nBlocks, (int)-2);

	// get the particle IDs for each cell
	getParticleIds_kernel <<<nBlocks, nFish >>>(pStartIds_dev, pEndIds_dev, cellIds_dev, cummCount_dev, gridCount_dev, nFish, nBlocks);

//	print_devArray(cellIds_dev, nFish);
//	print_devArray(pStartIds_dev, nCells);
//	print_devArray(pEndIds_dev, nCells);

    // execute the movement kernel
	int sharedMemSize = nFish * (sizeof(float2) + sizeof(float2) + sizeof(float));
    movement_kernel <<< nBlocks, nFish, sharedMemSize >>>(	pos_dev, vel_dev, Rs_dev, 
    														cellIds_dev, sortedIds_dev,
    														pStartIds_dev, pEndIds_dev, nCellsX, nCells,
    														nFish, //dev_params,
    														dev_XWstates, NULL, NULL);
}

// Wrapper for state setup kernel 
void launch_rngStateSetup_kernel(int * rng_blockSeeds, hiprandState * rngStates){
	rngStateSetup_kernel <<< nBlocks, nFish >>> (rng_blockSeeds, rngStates);
}

#endif // #ifndef _KERNEL_H_


